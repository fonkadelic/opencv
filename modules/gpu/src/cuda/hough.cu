#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or bpied warranties, including, but not limited to, the bpied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#include <thrust/sort.h>
#include "opencv2/gpu/device/common.hpp"
#include "opencv2/gpu/device/emulation.hpp"

namespace cv { namespace gpu { namespace device
{
    namespace hough
    {
        __device__ int g_counter;

        ////////////////////////////////////////////////////////////////////////
        // buildPointList

        const int PIXELS_PER_THREAD = 16;

        __global__ void buildPointList(const DevMem2Db src, unsigned int* list)
        {
            __shared__ int s_queues[4][32 * PIXELS_PER_THREAD];
            __shared__ int s_qsize[4];
            __shared__ int s_globStart[4];

            const int x = blockIdx.x * blockDim.x * PIXELS_PER_THREAD + threadIdx.x;
            const int y = blockIdx.y * blockDim.y + threadIdx.y;

            if (y >= src.rows)
                return;

            if (threadIdx.x == 0)
                s_qsize[threadIdx.y] = 0;

            __syncthreads();

            // fill the queue
            const uchar* srcRow = src.ptr(y);
            for (int i = 0, xx = x; i < PIXELS_PER_THREAD && xx < src.cols; ++i, xx += blockDim.x)
            {
                if (srcRow[xx])
                {
                    const unsigned int val = (y << 16) | xx;
                    const int qidx = Emulation::smem::atomicAdd(&s_qsize[threadIdx.y], 1);
                    s_queues[threadIdx.y][qidx] = val;
                }
            }

            __syncthreads();

            // let one thread reserve the space required in the global list
            if (threadIdx.x == 0 && threadIdx.y == 0)
            {
                // find how many items are stored in each list
                int totalSize = 0;
                for (int i = 0; i < blockDim.y; ++i)
                {
                    s_globStart[i] = totalSize;
                    totalSize += s_qsize[i];
                }

                // calculate the offset in the global list
                const int globalOffset = atomicAdd(&g_counter, totalSize);
                for (int i = 0; i < blockDim.y; ++i)
                    s_globStart[i] += globalOffset;
            }

            __syncthreads();

            // copy local queues to global queue
            const int qsize = s_qsize[threadIdx.y];
            int gidx = s_globStart[threadIdx.y] + threadIdx.x;
            for(int i = threadIdx.x; i < qsize; i += blockDim.x, gidx += blockDim.x)
                list[gidx] = s_queues[threadIdx.y][i];
        }

        int buildPointList_gpu(DevMem2Db src, unsigned int* list)
        {
            void* counterPtr;
            cudaSafeCall( hipGetSymbolAddress(&counterPtr, HIP_SYMBOL(g_counter)) );

            cudaSafeCall( hipMemset(counterPtr, 0, sizeof(int)) );

            const dim3 block(32, 4);
            const dim3 grid(divUp(src.cols, block.x * PIXELS_PER_THREAD), divUp(src.rows, block.y));

            cudaSafeCall( hipFuncSetCacheConfig(reinterpret_cast<const void*>(buildPointList), hipFuncCachePreferShared) );

            buildPointList<<<grid, block>>>(src, list);
            cudaSafeCall( hipGetLastError() );

            cudaSafeCall( hipDeviceSynchronize() );

            int totalCount;
            cudaSafeCall( hipMemcpy(&totalCount, counterPtr, sizeof(int), hipMemcpyDeviceToHost) );

            return totalCount;
        }

        ////////////////////////////////////////////////////////////////////////
        // linesAccum

        __global__ void linesAccumGlobal(const unsigned int* list, const int count, PtrStepi accum, const float irho, const float theta, const int numrho)
        {
            const int n = blockIdx.x;
            const float ang = n * theta;

            float sinVal;
            float cosVal;
            sincosf(ang, &sinVal, &cosVal);
            sinVal *= irho;
            cosVal *= irho;

            const int shift = (numrho - 1) / 2;

            int* accumRow = accum.ptr(n + 1);
            for (int i = threadIdx.x; i < count; i += blockDim.x)
            {
                const unsigned int val = list[i];

                const int x = (val & 0xFFFF);
                const int y = (val >> 16) & 0xFFFF;

                int r = __float2int_rn(x * cosVal + y * sinVal);
                r += shift;

                ::atomicAdd(accumRow + r + 1, 1);
            }
        }

        __global__ void linesAccumShared(const unsigned int* list, const int count, PtrStepi accum, const float irho, const float theta, const int numrho)
        {
            extern __shared__ int smem[];

            for (int i = threadIdx.x; i < numrho + 1; i += blockDim.x)
                smem[i] = 0;

            __syncthreads();

            const int n = blockIdx.x;
            const float ang = n * theta;

            float sinVal;
            float cosVal;
            sincosf(ang, &sinVal, &cosVal);
            sinVal *= irho;
            cosVal *= irho;

            const int shift = (numrho - 1) / 2;

            for (int i = threadIdx.x; i < count; i += blockDim.x)
            {
                const unsigned int val = list[i];

                const int x = (val & 0xFFFF);
                const int y = (val >> 16) & 0xFFFF;

                int r = __float2int_rn(x * cosVal + y * sinVal);
                r += shift;

                Emulation::smem::atomicAdd(&smem[r + 1], 1);
            }

            __syncthreads();

            int* accumRow = accum.ptr(n + 1);
            for (int i = threadIdx.x; i < numrho + 1; i += blockDim.x)
                accumRow[i] = smem[i];
        }

        void linesAccum_gpu(const unsigned int* list, int count, DevMem2Di accum, float rho, float theta, size_t sharedMemPerBlock, bool has20)
        {
            const dim3 block(has20 ? 1024 : 512);
            const dim3 grid(accum.rows - 2);

            cudaSafeCall( hipFuncSetCacheConfig(reinterpret_cast<const void*>(linesAccumShared), hipFuncCachePreferShared) );

            size_t smemSize = (accum.cols - 1) * sizeof(int);

            if (smemSize < sharedMemPerBlock - 1000)
                linesAccumShared<<<grid, block, smemSize>>>(list, count, accum, 1.0f / rho, theta, accum.cols - 2);
            else
                linesAccumGlobal<<<grid, block>>>(list, count, accum, 1.0f / rho, theta, accum.cols - 2);

            cudaSafeCall( hipGetLastError() );

            cudaSafeCall( hipDeviceSynchronize() );
        }

        ////////////////////////////////////////////////////////////////////////
        // linesGetResult

        __global__ void linesGetResult(const DevMem2Di accum, float2* out, int* votes, const int maxSize, const float rho, const float theta, const float threshold, const int numrho)
        {
            __shared__ int smem[8][32];

            const int x = blockIdx.x * (blockDim.x - 2) + threadIdx.x;
            const int y = blockIdx.y * (blockDim.y - 2) + threadIdx.y;

            if (x >= accum.cols || y >= accum.rows)
                return;

            smem[threadIdx.y][threadIdx.x] = accum(y, x);
            __syncthreads();

            const int r = x - 1;
            const int n = y - 1;

            if (threadIdx.x == 0 || threadIdx.x == blockDim.x - 1 || threadIdx.y == 0 || threadIdx.y == blockDim.y - 1 || r >= accum.cols - 2 || n >= accum.rows - 2)
                return;

            if (smem[threadIdx.y][threadIdx.x] > threshold &&
                smem[threadIdx.y][threadIdx.x] >  smem[threadIdx.y - 1][threadIdx.x] &&
                smem[threadIdx.y][threadIdx.x] >= smem[threadIdx.y + 1][threadIdx.x] &&
                smem[threadIdx.y][threadIdx.x] >  smem[threadIdx.y][threadIdx.x - 1] &&
                smem[threadIdx.y][threadIdx.x] >= smem[threadIdx.y][threadIdx.x + 1])
            {
                const float radius = (r - (numrho - 1) * 0.5f) * rho;
                const float angle = n * theta;

                const int ind = ::atomicAdd(&g_counter, 1);
                if (ind < maxSize)
                {
                    out[ind] = make_float2(radius, angle);
                    votes[ind] = smem[threadIdx.y][threadIdx.x];
                }
            }
        }

        int linesGetResult_gpu(DevMem2Di accum, float2* out, int* votes, int maxSize, float rho, float theta, float threshold, bool doSort)
        {
            void* counterPtr;
            cudaSafeCall( hipGetSymbolAddress(&counterPtr, HIP_SYMBOL(g_counter)) );

            cudaSafeCall( hipMemset(counterPtr, 0, sizeof(int)) );

            const dim3 block(32, 8);
            const dim3 grid(divUp(accum.cols, block.x - 2), divUp(accum.rows, block.y - 2));

            linesGetResult<<<grid, block>>>(accum, out, votes, maxSize, rho, theta, threshold, accum.cols - 2);
            cudaSafeCall( hipGetLastError() );

            cudaSafeCall( hipDeviceSynchronize() );

            int totalCount;
            cudaSafeCall( hipMemcpy(&totalCount, counterPtr, sizeof(int), hipMemcpyDeviceToHost) );

            totalCount = ::min(totalCount, maxSize);

            if (doSort && totalCount > 0)
            {
                thrust::device_ptr<float2> outPtr(out);
                thrust::device_ptr<int> votesPtr(votes);
                thrust::sort_by_key(votesPtr, votesPtr + totalCount, outPtr, thrust::greater<int>());
            }

            return totalCount;
        }
    }
}}}
